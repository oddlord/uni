#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""

#include <stdio.h>

#include "Image.h"
#include "PPM.h"

#include <cstdio>
#include <cassert>
#include <iostream>

#define _USE_MATH_DEFINES
#include <math.h>
#include <iomanip>

static void CheckCudaErrorAux(const char *, unsigned, const char *,
	hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

/**
* Check the return value of the CUDA runtime API call and exit
* the application if the call has failed.
*/
static void CheckCudaErrorAux(const char *file, unsigned line,
	const char *statement, hipError_t err) {
	if (err == hipSuccess)
		return;
	std::cerr << statement << " returned " << hipGetErrorString(err) << "("
		<< err << ") at " << file << ":" << line << std::endl;
	exit(1);
}

// Useful defines
#define NUMBER_THREAD_X 16
#define NUMBER_THREAD_Y 16
#define TILE_SIZE NUMBER_THREAD_X * NUMBER_THREAD_Y * 3 // each block matches with the input tile
#define clamp(x) (min(max((x), 0.0), 1.0))

// Global variables
const int maskRows = 5;
const int maskColumns = 5;
const int maskRowsRadius = maskRows / 2;
const int maskColumnsRadius = maskColumns / 2;
__constant__ float deviceMaskData[maskRows * maskColumns];


__global__ void convolution(float *I, float *P,
	int channels, int width, int height) {

	// Original columns/rows index before shifting
	int colOriginal = blockIdx.x * (blockDim.x - maskColumnsRadius*2) + threadIdx.x;
	int rowOriginal = blockIdx.y * (blockDim.y - maskRowsRadius*2) + threadIdx.y;

	// Thread columns and rows
	// (Original cols/rows shifted by the mask radius backwards)
	int colT = colOriginal - maskColumnsRadius;
	int rowT = rowOriginal - maskRowsRadius;

	int depth = threadIdx.z;

	// 1st phase: copy from global memory to shared memory (tiling)

	// As design choice, we assume that each block matches each input tile
	// meaning that each thread loads its own input pixel
	// but only the central ones computes the output pixel
	__shared__ float Ids[TILE_SIZE];
	int sharedMemoryPos = (threadIdx.y * blockDim.y + threadIdx.x)*channels + depth;
	
	// Actual tiling
	if (rowT >= 0 && rowT < height && colT >= 0 && colT < width) {
		Ids[sharedMemoryPos] = I[(rowT * width + colT) * channels + depth];
	}
	else { // check for ghost elements
		Ids[sharedMemoryPos] = 0.0f;
	}
	
	// Wait for other threads in the same block
	__syncthreads();

	// 2nd phase: evaluate convolution

	// This first IF is to check whether we're still inside the image boundaries or not
	if (rowT >= 0 && rowT < height && colT >= 0 && colT < width) {
		// This second IF is to check whether we're inside the central block area or not (border threads do not compute anything)
		if (threadIdx.x >= maskColumnsRadius && threadIdx.x < (blockDim.x - 2) && threadIdx.y >= maskRowsRadius && threadIdx.y < (blockDim.y - 2)) {
			float pValue = 0;

			int startCol = threadIdx.x - maskColumnsRadius;
			int startRow = threadIdx.y - maskRowsRadius;

			for (int i = 0; i < maskRows; i++) {
				for (int j = 0; j < maskColumns; j++) {
					int currentCol = startCol + j;
					int currentRow = startRow + i;

					// Check for ghost elements already done during tiling
					float iValue = Ids[(currentRow * blockDim.y + currentCol) * channels + depth];

					pValue += iValue * deviceMaskData[i * maskRows + j];
				}
			}

			// Store the result inside the output vector P in the global memory
			P[(rowT * width + colT) * channels + depth] = pValue;
		}
	}
}

__global__ void convolutionNoTiling(float *I, float *P,
	int channels, int width, int height) {

	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int depth = threadIdx.z;

	if (col < width && row < height && depth < channels) {

		// Evaluate convolution
		float pValue = 0;

		int startRow = row - maskRowsRadius;
		int startCol = col - maskColumnsRadius;

		for (int i = 0; i < maskRows; i++) {
			for (int j = 0; j < maskColumns; j++) {
				int currentRow = startRow + i;
				int currentCol = startCol + j;

				float iValue;

				// Check for ghost elements
				if (currentRow >= 0 && currentRow < height && currentCol >= 0 && currentCol < width) {
					iValue = I[(currentRow * width + currentCol) * channels + depth];
				}
				else {
					iValue = 0.0f;
				}

				pValue += iValue * deviceMaskData[i * maskRows + j];
			}
		}

		// Salva il risultato dal registro alla global
		P[(row * width + col) * channels + depth] = pValue;
	}
}


// Simple test to read/write PPM images, and process Image_t data
void test_images() {
	Image_t* inputImg = PPM_import("computer_programming.ppm");
	for (int i = 0; i < 300; i++) {
		Image_setPixel(inputImg, i, 100, 0, float(i) / 300);
		Image_setPixel(inputImg, i, 100, 1, float(i) / 300);
		Image_setPixel(inputImg, i, 100, 2, float(i) / 200);
	}
	PPM_export("test_output.ppm", inputImg);
	Image_t* newImg = PPM_import("test_output.ppm");
	inputImg = PPM_import("computer_programming.ppm");
	if (Image_is_same(inputImg, newImg))
		printf("Img uguali\n");
	else
		printf("Img diverse\n");
}

void constantFilter(float mask[])
{
	for (int x = 0; x < 5; x++)
	{
		for (int y = 0; y < 5; y++)
		{
			mask[x*5 + y] = (float)1 / (float)(5*5);
		}
	}
}

void identityFilter(float mask[])
{
	for (int x = 0; x < 5; x++)
	{
		for (int y = 0; y < 5; y++)
		{
			mask[x*5 + y] = 0;
		}
	}
	mask[2*5 + 2] = 1;
}

void gaussianFilter(float mask[], float sigma)
{
	// Means on X and Y are fixed to 0
	// Correlation coefficient is fixed to 0
	// Standard deviation (for both X and Y) is passed as parameter
	float r, s = 2.0 * sigma * sigma;

	// Sum is for normalization
	float sum = 0.0;

	// Generate 5x5 mask values
	for (int x = -2; x <= 2; x++)
	{
		for (int y = -2; y <= 2; y++)
		{
			r = sqrt(x*x + y*y);
			mask[(x + 2)*5 + (y + 2)] = (exp(-(r*r) / s)) / (M_PI * s);
			sum += mask[(x + 2) * 5 + (y + 2)];
		}
	}

	// Normalize the mask
	for (int i = 0; i < 5; ++i)
		for (int j = 0; j < 5; ++j)
			mask[i*5 + j] /= sum;

}

void inverseGaussianFilter(float mask[], float sigma)
{
	gaussianFilter(mask, sigma);

	float sum = 0.0;

	// Invert the mask
	for (int i = 0; i < 5; ++i) {
		for (int j = 0; j < 5; ++j) {
			mask[i * 5 + j] = 1 - mask[i * 5 + j];
			sum += mask[i * 5 + j];
		}
	}

	// Normalize the mask
	for (int i = 0; i < 5; ++i)
		for (int j = 0; j < 5; ++j)
			mask[i * 5 + j] /= sum;

}

void printFilter(float mask[]) {
	for (int i = 0; i < 5; ++i)
	{
		for (int j = 0; j < 5; ++j)
			std::cout << mask[i*5 + j] << "\t";
		std::cout << std::endl;
	}
	std::cin.ignore();
}

int main() {

	int imageChannels;
	int imageWidth;
	int imageHeight;
	Image_t* inputImage;
	Image_t* outputImage;
	float *hostInputImageData;
	float *hostOutputImageData;
	float *deviceInputImageData;
	float *deviceOutputImageData;
	
	// Mask matrix creation
	float hostMaskData[maskRows * maskColumns];
	gaussianFilter(hostMaskData, 1);
	// printFilter(hostMaskData); // uncomment to check mask values

	inputImage = PPM_import("computer_programming.ppm");

	assert(maskRows == 5); /* mask height is fixed to 5 in this exercise */
	assert(maskColumns == 5); /* mask width is fixed to 5 in this exercise */

	imageWidth = Image_getWidth(inputImage);
	imageHeight = Image_getHeight(inputImage);
	imageChannels = Image_getChannels(inputImage);

	outputImage = Image_new(imageWidth, imageHeight, imageChannels);

	hostInputImageData = Image_getData(inputImage);
	hostOutputImageData = Image_getData(outputImage);

	// Allocate device buffers
	CUDA_CHECK_RETURN(
		hipMalloc((void **)&deviceInputImageData,
			sizeof(float) * imageWidth * imageHeight * imageChannels));

	CUDA_CHECK_RETURN(
		hipMalloc((void **)&deviceOutputImageData,
			sizeof(float) * imageWidth * imageHeight * imageChannels));

	// Copy memory from host to device
	CUDA_CHECK_RETURN(
		hipMemcpyToSymbol(HIP_SYMBOL(deviceMaskData), hostMaskData, maskRows * maskColumns * sizeof(float)));
	CUDA_CHECK_RETURN(
		hipMemcpy(deviceInputImageData, hostInputImageData, sizeof(float) * imageWidth * imageHeight * imageChannels,
			hipMemcpyHostToDevice));

	// Evaluate block and thread number

	// # of blocks needed WITHOUT tiling is computed as
	// width (or height) of the image / width (or height) of a single block
	float numberBlockXNoTiling = (float)imageWidth / NUMBER_THREAD_X;
	float numberBlockYNoTiling = (float)imageHeight / NUMBER_THREAD_Y;

	// # of blocks needed WITH tiling is computed as
	// width (or height) of the image / width (or height) of an output tile
	// (this is due the fact that we chose to match blocks with input tiles)
	float numberBlockXTiling = (float)imageWidth / (NUMBER_THREAD_X - maskColumnsRadius*2);
	float numberBlockYTiling = (float)imageHeight / (NUMBER_THREAD_Y - maskRowsRadius*2);

	// the actual # of blocks is obtained by rounding up the previous value
	// use here # of block with/without tiling depending on which kernel is called
	int numberBlockX = ceil(numberBlockXTiling);
	int numberBlockY = ceil(numberBlockYTiling);
		
	dim3 dimGrid(numberBlockX, numberBlockY);
	dim3 dimBlock(NUMBER_THREAD_X, NUMBER_THREAD_Y, 3);
	convolution<<<dimGrid, dimBlock>>>(deviceInputImageData,
	 deviceOutputImageData, imageChannels, imageWidth, imageHeight);

	// Copy from device to host memory
	CUDA_CHECK_RETURN(
		hipMemcpy(hostOutputImageData, deviceOutputImageData, sizeof(float) * imageWidth * imageHeight * imageChannels,
			hipMemcpyDeviceToHost));

	PPM_export("processed_computer_programming.ppm", outputImage);

	// Free device memory
	// deviceMaskData memory doesn't need to be freed since it's a global variable
	hipFree(deviceInputImageData);
	hipFree(deviceOutputImageData);

	Image_delete(outputImage);
	Image_delete(inputImage);

	return 0;
}

